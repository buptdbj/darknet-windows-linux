#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")  
#endif

extern "C" {
#include "convolutional_layer.h"
#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    binary[i] = (x[i] >= 0) ? 1 : -1;
}

void binarize_gpu(float *x, int n, float *binary)
{
    binarize_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, binary);
    check_error(hipPeekAtLastError());
}

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < n; ++i){
        mean += fabs(input[i*size + s]);
    }
    mean = mean / n;
    for(i = 0; i < n; ++i){
        binary[i*size + s] = (input[i*size + s] > 0) ? mean : -mean;
    }
}

void binarize_input_gpu(float *input, int n, int size, float *binary)
{
    binarize_input_kernel<<<cuda_gridsize(size), BLOCK>>>(input, n, size, binary);
    check_error(hipPeekAtLastError());
}


__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < size; ++i){
        mean += fabs(weights[f*size + i]);
    }
    mean = mean / size;
    for(i = 0; i < size; ++i){
        binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
        //binary[f*size + i] = weights[f*size + i];
    }
}

void binarize_weights_gpu(float *weights, int n, int size, float *binary)
{
    binarize_weights_kernel<<<cuda_gridsize(n), BLOCK>>>(weights, n, size, binary);
    check_error(hipPeekAtLastError());
}

__global__ void cuda_f32_to_f16(float* input_f32, size_t size, half *output_f16)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) output_f16[idx] = __float2half(input_f32[idx]);
    //if (idx < size) *((unsigned short *)output_f16 + idx) = __float2half(input_f32[idx]);
}

void cuda_convert_f32_to_f16(float* input_f32, size_t size, float *output_f16) {
    cuda_f32_to_f16 <<< size / BLOCK + 1, BLOCK, 0, get_cuda_stream() >>> (input_f32, size, (half *)output_f16);
}

__global__ void cuda_f16_to_f32(half* input_f16, size_t size, float *output_f32)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) output_f32[idx] = __half2float(input_f16[idx]);
    //if (idx < size) output_f32[idx] = __half2float(*((unsigned short *)input_f16 + idx));
}

void cuda_convert_f16_to_f32(float* input_f16, size_t size, float *output_f32) {
    cuda_f16_to_f32 <<< size / BLOCK + 1, BLOCK, 0, get_cuda_stream() >>> ((half *)input_f16, size, output_f32);
}

half *cuda_make_f16_from_f32_array(float *src, size_t n)
{
    half *dst16;
    size_t size = sizeof(half)*n;
    check_error(hipMalloc((void **)&dst16, size));
    if (src) {
        cuda_convert_f32_to_f16(src, n, (float *)dst16);
    }
    if (!dst16) error("Cuda malloc failed\n");
    return dst16;
}

void forward_convolutional_layer_gpu(convolutional_layer l, network_state state)
{
    fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);
    if(l.binary){
        binarize_weights_gpu(l.weights_gpu, l.n, l.c*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
    }

    if(l.xnor){
        binarize_weights_gpu(l.weights_gpu, l.n, l.c*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
        binarize_gpu(state.input, l.c*l.h*l.w*l.batch, l.binary_input_gpu);
        state.input = l.binary_input_gpu;
    }

#ifdef CUDNN
    float one = 1;    // alpha[0], beta[0] is float for HALF and FLOAT
    float alpha = 1, beta = 0; 

#ifdef CUDNN_HALF
    // Note: For improved performance it is advised to use beta[0] = 0.0. 
    // For Tensor Core: hipdnnSetConvolutionMathType() where hipdnnMathType_t mathType = HIPDNN_TENSOR_OP_MATH;
    // 1. or HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM and use HIPDNN_DATA_HALF
    // 2. or HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED
    // More: http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#tensor_ops

    const size_t input16_size = l.batch*l.c*l.w*l.h;
    const size_t output16_size = l.batch*l.out_c*l.out_h*l.out_w;

    if (*state.net.max_input16_size < input16_size) {
        //printf("\n input16_size: cur = %zu \t max = %zu \n", input16_size, *state.net.max_input16_size);
        *state.net.max_input16_size = input16_size;
        if (*state.net.input16_gpu) cuda_free(*state.net.input16_gpu);
        *state.net.input16_gpu = (float *)cuda_make_f16_from_f32_array(NULL, *state.net.max_input16_size);
    }
    float *input16 = *state.net.input16_gpu;

    if (*state.net.max_output16_size < output16_size) {
        *state.net.max_output16_size = output16_size;
        if (*state.net.output16_gpu) cuda_free(*state.net.output16_gpu);
        *state.net.output16_gpu = (float *)cuda_make_f16_from_f32_array(NULL, *state.net.max_output16_size);
    }
    float *output16 = *state.net.output16_gpu;

    cuda_convert_f32_to_f16(state.input, input16_size, input16);

    //fill_ongpu(output16_size / 2, 0, (float *)output16, 1);
    hipdnnConvolutionForward(cudnn_handle(),
        &alpha,
        l.srcTensorDesc,
        input16,
        l.weightDesc,
        l.weights_gpu16,
        l.convDesc,
        l.fw_algo,
        state.workspace,
        l.workspace_size,
        &beta,
        l.dstTensorDesc,
        output16);
    

    if (l.batch_normalize) 
    {        
        if (state.train) // Training
        {
            copy_ongpu(l.outputs*l.batch / 2, output16, 1, l.x_gpu, 1);
            //hipMemcpyAsync(l.x_gpu, output16, l.outputs*l.batch*sizeof(half), hipMemcpyDefault, get_cuda_stream());
            float one = 1;
            float zero = 0;
            // Batch-normalization can still take FP16 inputs and outputs, saving half the bandwidth
            // compared to FP32, it�s just that the statistics and value adjustment should be done in FP32.
            hipdnnBatchNormalizationForwardTraining(cudnn_handle(),
                HIPDNN_BATCHNORM_SPATIAL,
                &one,
                &zero,
                l.normDstTensorDescF16,
                l.x_gpu,            // input
                l.normDstTensorDescF16,
                output16,            // output
                l.normTensorDesc,
                l.scales_gpu,
                l.biases_gpu,
                .01,
                l.rolling_mean_gpu,        // output (should be FP32)
                l.rolling_variance_gpu,    // output (should be FP32)
                .00001,
                l.mean_gpu,            // output (should be FP32)
                l.variance_gpu);    // output (should be FP32)

            cuda_convert_f16_to_f32(output16, output16_size, l.output_gpu);
            //forward_batchnorm_layer_gpu(l, state);
        }
        else // Detection
        {
            cuda_convert_f16_to_f32(output16, output16_size, l.output_gpu);
            normalize_gpu(l.output_gpu, l.rolling_mean_gpu, l.rolling_variance_gpu, l.batch, l.out_c, l.out_h*l.out_w);
            scale_bias_gpu(l.output_gpu, l.scales_gpu, l.batch, l.out_c, l.out_h*l.out_w);
            add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.out_c, l.out_w*l.out_h);
        }
    }
    else // BIAS only
    {
        cuda_convert_f16_to_f32(output16, output16_size, l.output_gpu);
        add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }    

#else

    hipdnnConvolutionForward(cudnn_handle(),
                &one,
                l.srcTensorDesc,
                state.input,
                l.weightDesc,
                l.weights_gpu,
                l.convDesc,
                l.fw_algo,
                state.workspace,
                l.workspace_size,
                &one,
                l.dstTensorDesc,
                l.output_gpu);
#endif    // CUDNN_HALF


#else
    int i;
    int m = l.n;
    int k = l.size*l.size*l.c;
    int n = l.out_w*l.out_h;
    for(i = 0; i < l.batch; ++i){
        im2col_ongpu(state.input + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.workspace);
        float * a = l.weights_gpu;
        float * b = state.workspace;
        float * c = l.output_gpu;
        gemm_ongpu(0,0,m,n,k,1.,a,k,b,n,1.,c+i*m*n,n);
    }
#endif

#ifndef CUDNN_HALF
    if (l.batch_normalize) {
        forward_batchnorm_layer_gpu(l, state);
    }
    else {
        add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
#endif // no CUDNN_HALF

    activate_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation);
    //if(l.dot > 0) dot_error_gpu(l);
    if(l.binary || l.xnor) swap_binary(&l);
    //hipDeviceSynchronize();    // for correct profiling of performance
}

void backward_convolutional_layer_gpu(convolutional_layer l, network_state state)
{
    gradient_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);

    backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);

#ifndef CUDNN_HALF
    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, state);
    } else {
        //backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
#endif // no CUDNN_HALF
    float *original_input = state.input;

    if(l.xnor) state.input = l.binary_input_gpu;
#ifdef CUDNN
    float one = 1;
    float alpha = 1, beta = 0;

#ifdef CUDNN_HALF
        
    const size_t input16_size = l.batch*l.c*l.w*l.h;
    const size_t delta16_size = l.batch*l.n*l.out_w*l.out_h;
    
    if (*state.net.max_input16_size < input16_size) {        
        *state.net.max_input16_size = input16_size;
        if(*state.net.input16_gpu) cuda_free(*state.net.input16_gpu);
        *state.net.input16_gpu = (float *)cuda_make_f16_from_f32_array(NULL, *state.net.max_input16_size);
    }
    float *input16 = *state.net.input16_gpu;

    if (*state.net.max_output16_size < delta16_size) {
        *state.net.max_output16_size = delta16_size;
        if(*state.net.output16_gpu) cuda_free(*state.net.output16_gpu);
        *state.net.output16_gpu = (float *)cuda_make_f16_from_f32_array(NULL, *state.net.max_output16_size);
    }
    float *delta16 = *state.net.output16_gpu;

    cuda_convert_f32_to_f16(state.input, input16_size, input16);
    cuda_convert_f32_to_f16(l.delta_gpu, delta16_size, delta16);

    if (l.batch_normalize) {
        //if (!state.train) {
        //    l.mean_gpu = l.rolling_mean_gpu;
        //    l.variance_gpu = l.rolling_variance_gpu;
        //}
        float one = 1;
        float zero = 0;
        hipdnnBatchNormalizationBackward(cudnn_handle(),
            HIPDNN_BATCHNORM_SPATIAL,
            &one,
            &zero,
            &one,
            &one,
            l.normDstTensorDescF16,
            l.x_gpu,                // input
            l.normDstTensorDescF16,
            delta16,                // input
            l.normDstTensorDescF16,
            l.x_norm_gpu,            // output
            l.normTensorDesc,
            l.scales_gpu,            // output (should be FP32)
            l.scale_updates_gpu,    // output (should be FP32)
            l.bias_updates_gpu,        // output (should be FP32)
            .00001,
            l.mean_gpu,                // input (should be FP32)
            l.variance_gpu);        // input (should be FP32)
        copy_ongpu(l.outputs*l.batch / 2, l.x_norm_gpu, 1, delta16, 1);
        //hipMemcpyAsync(delta16, l.x_norm_gpu, l.outputs*l.batch * sizeof(half), hipMemcpyDefault, get_cuda_stream());
    }
    else
    {
        //backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
    }

    // convert input: state.input (x), l.delta_gpu (y) from fp32 to fp16
    // get output: l.weight_updates_gpu (dw) and convert it to fp32 (ONLY if it is fp16)

    // calculate conv weight updates
    // Already: l.weight_updates_gpu = (l.weight_updates_gpu - l.weight*decay*batch*subdivision)*momentum
    //   so we should copy f32 to f16, or compute: f16=(w_up - w*d*b*s)*m
    cuda_convert_f32_to_f16(l.weight_updates_gpu, l.c*l.n*l.size*l.size, l.weight_updates_gpu16);

    hipdnnConvolutionBackwardFilter(cudnn_handle(),
        &one,
        l.srcTensorDesc,
        input16, //state.input,
        l.ddstTensorDesc,
        delta16, //l.delta_gpu,
        l.convDesc,
        l.bf_algo,
        state.workspace,
        l.workspace_size,
        &one,
        l.dweightDesc,
        l.weight_updates_gpu16);    // l.weight_updates_gpu);

    cuda_convert_f16_to_f32(l.weight_updates_gpu16, l.c*l.n*l.size*l.size, l.weight_updates_gpu);

    if (state.delta) {
        if (l.binary || l.xnor) swap_binary(&l);

        // http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#hipdnnConvolutionBackwardData
        // calculate delta for the next layer
        // convert input: l.weights_gpu (w), l.delta_gpu (dy) from fp32 to fp16
        // get output: state.delta (dx) and convert it to fp32 (ONLY if it is fp16)    
        hipdnnConvolutionBackwardData(cudnn_handle(),
            &alpha,
            l.weightDesc,
            l.weights_gpu16, //l.weights_gpu,
            l.ddstTensorDesc,
            delta16, //l.delta_gpu,
            l.convDesc,
            l.bd_algo,
            state.workspace,
            l.workspace_size,
            &beta,
            l.dsrcTensorDesc,
            input16);    // state.delta);

        cuda_convert_f16_to_f32(input16, input16_size, state.delta);

        if (l.binary || l.xnor) swap_binary(&l);
        if (l.xnor) gradient_array_ongpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, state.delta);
    }
#else    // CUDNN_HALF

    // calculate conv weight updates
    // if used: beta=1 then loss decreases faster
    hipdnnConvolutionBackwardFilter(cudnn_handle(),
            &one,
            l.srcTensorDesc,
            state.input,
            l.ddstTensorDesc,
            l.delta_gpu,
            l.convDesc,
            l.bf_algo,
            state.workspace,
            l.workspace_size,
            &one,
            l.dweightDesc,
            l.weight_updates_gpu);

    if(state.delta){
        if(l.binary || l.xnor) swap_binary(&l);
        // http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#hipdnnConvolutionBackwardData
        // calculate delta for the next layer
        hipdnnConvolutionBackwardData(cudnn_handle(),
                &one,
                l.weightDesc,
                l.weights_gpu,
                l.ddstTensorDesc,
                l.delta_gpu,
                l.convDesc,
                l.bd_algo,
                state.workspace,
                l.workspace_size,
                &one,
                l.dsrcTensorDesc,
                state.delta);
        if(l.binary || l.xnor) swap_binary(&l);
        if(l.xnor) gradient_array_ongpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, state.delta);
    }

#endif    // CUDNN_HALF

#else    // CUDNN
    int m = l.n;
    int n = l.size*l.size*l.c;
    int k = l.out_w*l.out_h;

    int i;
    for(i = 0; i < l.batch; ++i){
        float * a = l.delta_gpu;
        float * b = state.workspace;
        float * c = l.weight_updates_gpu;

        im2col_ongpu(state.input + i*l.c*l.h*l.w, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.workspace);
        gemm_ongpu(0,1,m,n,k,1,a + i*m*k,k,b,k,1,c,n);

        if(state.delta){
            if(l.binary || l.xnor) swap_binary(&l);
            float * a = l.weights_gpu;
            float * b = l.delta_gpu;
            float * c = state.workspace;

            gemm_ongpu(1,0,n,k,m,1,a,n,b + i*k*m,k,0,c,k);

            col2im_ongpu(state.workspace, l.c,  l.h,  l.w,  l.size,  l.stride, l.pad, state.delta + i*l.c*l.h*l.w);
            if(l.binary || l.xnor) {
                swap_binary(&l);
            }
            if(l.xnor) gradient_array_ongpu(original_input + i*l.c*l.h*l.w, l.c*l.h*l.w, HARDTAN, state.delta + i*l.c*l.h*l.w);
        }
    }
#endif
}

void pull_convolutional_layer(convolutional_layer layer)
{
    cuda_pull_array(layer.weights_gpu, layer.weights, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_pull_array(layer.weight_updates_gpu, layer.weight_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_pull_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_pull_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_pull_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_pull_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }
    if (layer.adam){
        cuda_pull_array(layer.m_gpu, layer.m, layer.c*layer.n*layer.size*layer.size);
        cuda_pull_array(layer.v_gpu, layer.v, layer.c*layer.n*layer.size*layer.size);
    }
}

void push_convolutional_layer(convolutional_layer layer)
{
    cuda_push_array(layer.weights_gpu, layer.weights, layer.c*layer.n*layer.size*layer.size);
#ifdef CUDNN_HALF
    cuda_convert_f32_to_f16(layer.weights_gpu, layer.c*layer.n*layer.size*layer.size, layer.weights_gpu16);
#endif
    cuda_push_array(layer.biases_gpu, layer.biases, layer.n);
    cuda_push_array(layer.weight_updates_gpu, layer.weight_updates, layer.c*layer.n*layer.size*layer.size);
    cuda_push_array(layer.bias_updates_gpu, layer.bias_updates, layer.n);
    if (layer.batch_normalize){
        cuda_push_array(layer.scales_gpu, layer.scales, layer.n);
        cuda_push_array(layer.rolling_mean_gpu, layer.rolling_mean, layer.n);
        cuda_push_array(layer.rolling_variance_gpu, layer.rolling_variance, layer.n);
    }
    if (layer.adam){
        cuda_push_array(layer.m_gpu, layer.m, layer.c*layer.n*layer.size*layer.size);
        cuda_push_array(layer.v_gpu, layer.v, layer.c*layer.n*layer.size*layer.size);
    }
}

void update_convolutional_layer_gpu(convolutional_layer layer, int batch, float learning_rate, float momentum, float decay)
{
    int size = layer.size*layer.size*layer.c*layer.n;
    axpy_ongpu(layer.n, learning_rate/batch, layer.bias_updates_gpu, 1, layer.biases_gpu, 1);
    scal_ongpu(layer.n, momentum, layer.bias_updates_gpu, 1);

    if(layer.scales_gpu){
        axpy_ongpu(layer.n, learning_rate/batch, layer.scale_updates_gpu, 1, layer.scales_gpu, 1);
        scal_ongpu(layer.n, momentum, layer.scale_updates_gpu, 1);
    }

    if(layer.adam){
        scal_ongpu(size, layer.B1, layer.m_gpu, 1);
        scal_ongpu(size, layer.B2, layer.v_gpu, 1);

        axpy_ongpu(size, -decay*batch, layer.weights_gpu, 1, layer.weight_updates_gpu, 1);

        axpy_ongpu(size, -(1-layer.B1), layer.weight_updates_gpu, 1, layer.m_gpu, 1);
        mul_ongpu(size, layer.weight_updates_gpu, 1, layer.weight_updates_gpu, 1);
        axpy_ongpu(size, (1-layer.B2), layer.weight_updates_gpu, 1, layer.v_gpu, 1);

        adam_gpu(size, layer.weights_gpu, layer.m_gpu, layer.v_gpu, layer.B1, layer.B2, learning_rate/batch, layer.eps, layer.t+1);
        fill_ongpu(size, 0, layer.weight_updates_gpu, 1);
    }else{
        // update weights:
        // weights_gpu = weights_gpu*(1 - decay*lr) + weight_updates_gpu*lr / (batch*subdivision) =
        //  weights_gpu*(1 - 0.0005*0.001) + weight_updates_gpu*0.001/(64*8) = 
        //  weights_gpu * 0.999 999 5 + weight_updates_gpu * 0.000 001 953125
        // 
        // weight_updates_gpu = (weight_updates_gpu - weights_gpu*decay*batch*subdivision)*momentum = 
        //  (weight_updates_gpu - weights_gpu * 0.0005 * 64 * 8) * 0.9 = 
        //  weight_updates_gpu*0.9 - weights_gpu*0.2304
        axpy_ongpu(size, -decay*batch, layer.weights_gpu, 1, layer.weight_updates_gpu, 1);
        axpy_ongpu(size, learning_rate/batch, layer.weight_updates_gpu, 1, layer.weights_gpu, 1);
        scal_ongpu(size, momentum, layer.weight_updates_gpu, 1);
    }
}


